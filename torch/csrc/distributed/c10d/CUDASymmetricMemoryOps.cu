#include "hip/hip_runtime.h"
#if defined(CUDART_VERSION) && CUDART_VERSION >= 12030

#include <ATen/ATen.h>
#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>

#ifndef AT_PER_OPERATOR_HEADERS
#include <ATen/Functions.h>
#include <ATen/NativeFunctions.h>
#else
#include <ATen/ops/empty_like.h>
#endif

#include <torch/library.h>

#include <torch/csrc/distributed/c10d/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/CUDASymmetricMemory.hpp>

#define INT_SWITCH_CASE(name, val, ...) \
  case val: {                           \
    constexpr int name = val;           \
    __VA_ARGS__();                      \
    break;                              \
  }

#define DISPATCH_WORLD_SIZES(world_size, ...)      \
  switch (world_size) {                            \
    INT_SWITCH_CASE(k_world_size, 8, __VA_ARGS__); \
    INT_SWITCH_CASE(k_world_size, 4, __VA_ARGS__); \
    INT_SWITCH_CASE(k_world_size, 2, __VA_ARGS__); \
    default: {                                     \
      constexpr int k_world_size = -1;             \
      __VA_ARGS__();                               \
    }                                              \
  }

#define DISPATCH_ALIGNMENTS_16_8_4(alignment, ...)                    \
  switch (alignment) {                                                \
    INT_SWITCH_CASE(k_alignment, 16, __VA_ARGS__);                    \
    INT_SWITCH_CASE(k_alignment, 8, __VA_ARGS__);                     \
    INT_SWITCH_CASE(k_alignment, 4, __VA_ARGS__);                     \
    default: {                                                        \
      TORCH_CHECK(false, "Not implemented for aligment=", alignment); \
    }                                                                 \
  }

#define AT_DISPATCH_FLOAT_AND_BFLOAT16(scalar_type, name, ...)         \
  AT_DISPATCH_SWITCH(                                                  \
      scalar_type, name, AT_DISPATCH_CASE(at::kBFloat16, __VA_ARGS__); \
      AT_DISPATCH_CASE(at::kFloat, __VA_ARGS__));

namespace {

using namespace c10d::symmetric_memory;

size_t get_and_verify_alignment(const at::Tensor& input, const char* op_name) {
  const size_t min_alignment = std::max(4l, input.element_size());
  // Only check the offset since the multicast address is always at least
  // 128-bit aligned
  const size_t ptr_alignment = get_alignment(
      static_cast<size_t>(input.storage_offset() * input.element_size()));
  TORCH_CHECK(
      ptr_alignment >= min_alignment,
      op_name,
      "<",
      input.scalar_type(),
      ">: input ptr + offset must be at least ",
      min_alignment,
      "-byte aligned.");

  const size_t size_alignment =
      get_alignment(static_cast<size_t>(input.numel() * input.element_size()));
  TORCH_CHECK(
      size_alignment >= min_alignment,
      op_name,
      "<",
      input.scalar_type(),
      ">: input size must be at least ",
      min_alignment,
      "-byte aligned.");
  return std::min(ptr_alignment, size_alignment);
}

void init_elementwise_launch_config(
    size_t numel,
    size_t element_size,
    size_t alignment,
    size_t splits,
    size_t max_num_blocks,
    size_t max_num_threads,
    int& num_blocks,
    int& num_threads) {
  // Align to preserve alignment in each split
  const size_t aligned_numel = at::round_up(numel, alignment * splits);
  const size_t numel_per_split = aligned_numel / splits;
  const size_t numel_per_thread = alignment / element_size;

  if (numel_per_split <= max_num_threads * numel_per_thread) {
    num_blocks = 1;
    num_threads = at::round_up(
        at::ceil_div(numel_per_split, numel_per_thread),
        static_cast<size_t>(C10_WARP_SIZE));
  } else {
    num_blocks = std::min(
        at::ceil_div(numel_per_split, max_num_threads * numel_per_thread),
        max_num_blocks);
    num_threads = max_num_threads;
  }
}

template <typename T, int alignment>
static __global__ void multimem_all_reduce_kernel(
    T* input_mc_ptr,
    size_t numel,
    uint32_t** signal_pads,
    size_t rank,
    size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t numel_per_rank =
      at::round_up(numel, alignment * world_size) / world_size;
  const size_t start = numel_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    if (start + i >= numel) {
      continue;
    }
    auto vec = multimem_ld_reduce_add<alignment>(input_mc_ptr + start + i);
    multimem_st<alignment>(input_mc_ptr + start + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<MemOpSem::AcqRel>(signal_pads, rank, world_size);
}

at::Tensor multimem_all_reduce_(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(
      input.is_contiguous(), "multimem_all_reduce_: input must be contiguous.");
  TORCH_CHECK(
      reduce_op == "sum",
      "multimem_all_reduce_: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input);
  TORCH_CHECK(
      symm_mem != nullptr,
      "multimem_all_reduce_: input must be allocated with empty_strided_p2p().");
  TORCH_CHECK(
      symm_mem->has_multicast_support(),
      "multimem_all_reduce_: multicast support is required.");

  const size_t alignment =
      get_and_verify_alignment(input, "multimem_all_reduce_");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      symm_mem->get_world_size(),
      8,
      1024,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "multimem_all_reduce_", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          multimem_all_reduce_kernel<scalar_t, k_alignment>
              <<<num_blocks,
                 num_threads,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  reinterpret_cast<scalar_t*>(symm_mem->get_multicast_ptr()) +
                      input.storage_offset(),
                  input.numel(),
                  reinterpret_cast<uint32_t**>(
                      symm_mem->get_signal_pad_ptrs_dev()),
                  symm_mem->get_rank(),
                  symm_mem->get_world_size());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
      });
  return input;
}

template <typename T, int alignment>
static __global__ void multimem_one_shot_all_reduce_kernel(
    T* input_mc_ptr,
    T* output_ptr,
    size_t numel,
    uint32_t** signal_pads,
    size_t rank,
    size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel; i += stride) {
    auto vec = multimem_ld_reduce_add<alignment>(input_mc_ptr + i);
    st_vec<alignment>(output_ptr + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
}

at::Tensor multimem_one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(
      input.is_contiguous(),
      "multimem_one_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      out.is_contiguous(),
      "multimem_one_shot_all_reduce: output must be contiguous.");
  TORCH_CHECK(
      out.sizes() == input.sizes(),
      "multimem_one_shot_all_reduce: input/output size mismatch.");
  TORCH_CHECK(
      reduce_op == "sum",
      "multimem_one_shot_all_reduce: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input);
  TORCH_CHECK(
      symm_mem != nullptr,
      "multimem_one_shot_all_reduce: input must be allocated with empty_strided_p2p().");
  TORCH_CHECK(
      symm_mem->has_multicast_support(),
      "multimem_one_shot_all_reduce: requires multicast support.");

  const size_t alignment =
      get_and_verify_alignment(input, "multimem_one_shot_all_reduce");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      1,
      8,
      1024,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "multimem_one_shot_all_reduce", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          multimem_one_shot_all_reduce_kernel<scalar_t, k_alignment>
              <<<num_blocks,
                 num_threads,
                 0,
                 at::cuda::getCurrentCUDAStream()>>>(
                  reinterpret_cast<scalar_t*>(symm_mem->get_multicast_ptr()) +
                      input.storage_offset(),
                  out.data_ptr<scalar_t>(),
                  input.numel(),
                  reinterpret_cast<uint32_t**>(
                      symm_mem->get_signal_pad_ptrs_dev()),
                  symm_mem->get_rank(),
                  symm_mem->get_world_size());
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
      });
  return out;
}

at::Tensor multimem_one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  auto out = at::empty_like(input);
  return multimem_one_shot_all_reduce_out(input, reduce_op, group_name, out);
}

// One-shot all-reduce is register-intensive because it stages values loaded
// from peers in registers before performing reduction. Setting the thread
// count to 512 to prevent/alleviate register spill.
constexpr size_t one_shot_all_reduce_max_num_blocks = 8;
constexpr size_t one_shot_all_reduce_max_num_threads = 512;

template <typename T, int alignment, int k_world_size>
static __launch_bounds__(one_shot_all_reduce_max_num_threads) __global__
    void one_shot_all_reduce_kernel(
        T** input_ptrs,
        T* output_ptr,
        size_t input_offset,
        size_t numel,
        uint32_t** signal_pads,
        size_t rank,
        size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;

  for (size_t i = offset; i < numel; i += stride) {
    auto vec = load_and_reduce<T, alignment, k_world_size>(
        input_ptrs, rank, world_size, input_offset + i);
    st_vec<alignment>(output_ptr + i, vec);
  }

  __syncthreads();
  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
}

at::Tensor one_shot_all_reduce_out(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name,
    at::Tensor out) {
  TORCH_CHECK(
      input.is_contiguous(), "one_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      out.is_contiguous(), "one_shot_all_reduce: output must be contiguous.");
  TORCH_CHECK(
      out.sizes() == input.sizes(),
      "one_shot_all_reduce: input/output size mismatch.");
  TORCH_CHECK(
      reduce_op == "sum",
      "one_shot_all_reduce: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input);
  TORCH_CHECK(
      symm_mem != nullptr,
      "one_shot_all_reduce: input must be allocated with empty_strided_p2p().");

  const size_t alignment =
      get_and_verify_alignment(input, "one_shot_all_reduce");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      1,
      one_shot_all_reduce_max_num_blocks,
      one_shot_all_reduce_max_num_threads,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "one_shot_all_reduce", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          DISPATCH_WORLD_SIZES(symm_mem->get_world_size(), [&]() {
            one_shot_all_reduce_kernel<scalar_t, k_alignment, k_world_size>
                <<<num_blocks,
                   num_threads,
                   0,
                   at::cuda::getCurrentCUDAStream()>>>(
                    reinterpret_cast<scalar_t**>(
                        symm_mem->get_buffer_ptrs_dev()),
                    out.data_ptr<scalar_t>(),
                    input.storage_offset(),
                    input.numel(),
                    reinterpret_cast<uint32_t**>(
                        symm_mem->get_signal_pad_ptrs_dev()),
                    symm_mem->get_rank(),
                    symm_mem->get_world_size());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
        });
      });
  return out;
}

at::Tensor one_shot_all_reduce(
    const at::Tensor& input,
    std::string reduce_op,
    std::string group_name) {
  auto out = at::empty_like(input);
  return one_shot_all_reduce_out(input, reduce_op, group_name, out);
}

constexpr size_t two_shot_all_reduce_max_num_blocks = 24;
constexpr size_t two_shot_all_reduce_max_num_threads = 512;

template <typename T, int alignment, int k_world_size>
static __launch_bounds__(two_shot_all_reduce_max_num_threads) __global__
    void two_shot_all_reduce_kernel(
        T** input_ptrs,
        size_t input_offset,
        size_t numel,
        uint32_t** signal_pads,
        size_t rank,
        size_t world_size) {
  static_assert(alignment % sizeof(T) == 0);
  constexpr size_t numel_per_thread = alignment / sizeof(T);

  sync_remote_blocks<MemOpSem::Relaxed>(signal_pads, rank, world_size);
  __syncthreads();

  const size_t numel_per_rank =
      at::round_up(numel, alignment * world_size) / world_size;
  const size_t start = numel_per_rank * rank;

  auto offset = (blockDim.x * blockIdx.x + threadIdx.x) * numel_per_thread;
  auto stride = blockDim.x * gridDim.x * numel_per_thread;
  for (size_t i = offset; i < numel_per_rank; i += stride) {
    if (start + i >= numel) {
      continue;
    }
    auto vec = load_and_reduce<T, alignment, k_world_size>(
        input_ptrs, rank, world_size, input_offset + start + i);
    for (size_t step = 0; step < world_size; ++step) {
      size_t remote_rank = (rank + step) % world_size;
      st_vec<alignment>(
          input_ptrs[remote_rank] + input_offset + start + i, vec);
    }
  }

  __syncthreads();
  sync_remote_blocks<MemOpSem::AcqRel>(signal_pads, rank, world_size);
}

at::Tensor two_shot_all_reduce_(
    at::Tensor input,
    std::string reduce_op,
    std::string group_name) {
  TORCH_CHECK(
      input.is_contiguous(), "two_shot_all_reduce: input must be contiguous.");
  TORCH_CHECK(
      reduce_op == "sum",
      "two_shot_all_reduce: only sum is supported for now.");

  auto symm_mem = c10d::symmetric_memory::rendezvous(input);
  TORCH_CHECK(
      symm_mem != nullptr,
      "two_shot_all_reduce: input must be allocated with empty_strided_p2p().");

  const size_t alignment =
      get_and_verify_alignment(input, "two_shot_all_reduce");

  int num_blocks = 0, num_threads = 0;
  init_elementwise_launch_config(
      input.numel(),
      input.element_size(),
      alignment,
      symm_mem->get_world_size(),
      two_shot_all_reduce_max_num_blocks,
      two_shot_all_reduce_max_num_threads,
      num_blocks,
      num_threads);

  AT_DISPATCH_FLOAT_AND_BFLOAT16(
      input.scalar_type(), "two_shot_all_reduce", [&]() {
        DISPATCH_ALIGNMENTS_16_8_4(alignment, [&]() {
          DISPATCH_WORLD_SIZES(symm_mem->get_world_size(), [&]() {
            two_shot_all_reduce_kernel<scalar_t, k_alignment, k_world_size>
                <<<num_blocks,
                   num_threads,
                   0,
                   at::cuda::getCurrentCUDAStream()>>>(
                    reinterpret_cast<scalar_t**>(
                        symm_mem->get_buffer_ptrs_dev()),
                    input.storage_offset(),
                    input.numel(),
                    reinterpret_cast<uint32_t**>(
                        symm_mem->get_signal_pad_ptrs_dev()),
                    symm_mem->get_rank(),
                    symm_mem->get_world_size());
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
        });
      });
  return input;
}

TORCH_LIBRARY_FRAGMENT(symm_mem, m) {
  m.def(
      "multimem_all_reduce_(Tensor(a!) input, str reduce_op, str group_name) -> Tensor(a!)",
      torch::dispatch(c10::DispatchKey::CUDA, ::multimem_all_reduce_),
      {at::Tag::pt2_compliant_tag});

  // NOTE: [multimem_one_shot_all_reduce]
  // multimem.ld_reduce does not guarantee a fixed accumulation order. This
  // means that while multimem_one_shot_all_reduce is faster and has higher
  // numerical accuracy than one_shot_all_reduce, it doesn't guarantee
  // identical results across ranks. There may be use cases that can take
  // advantage of this property, but it should not be used without
  // understanding the caveats.
  m.def(
      "multimem_one_shot_all_reduce(Tensor input, str reduce_op, str group_name) -> Tensor",
      torch::dispatch(c10::DispatchKey::CUDA, ::multimem_one_shot_all_reduce),
      {at::Tag::pt2_compliant_tag});

  m.def(
      "multimem_one_shot_all_reduce_out(Tensor input, str reduce_op, str group_name, Tensor(a!) out) -> Tensor(a!)",
      torch::dispatch(
          c10::DispatchKey::CUDA, ::multimem_one_shot_all_reduce_out),
      {at::Tag::pt2_compliant_tag});

  m.def(
      "one_shot_all_reduce(Tensor input, str reduce_op, str group_name) -> Tensor",
      torch::dispatch(c10::DispatchKey::CUDA, ::one_shot_all_reduce),
      {at::Tag::pt2_compliant_tag});

  m.def(
      "one_shot_all_reduce_out(Tensor input, str reduce_op, str group_name, Tensor(a!) out) -> Tensor(a!)",
      torch::dispatch(c10::DispatchKey::CUDA, ::one_shot_all_reduce_out),
      {at::Tag::pt2_compliant_tag});

  m.def(
      "two_shot_all_reduce_(Tensor(a!) input, str reduce_op, str group_name) -> Tensor(a!)",
      torch::dispatch(c10::DispatchKey::CUDA, ::two_shot_all_reduce_),
      {at::Tag::pt2_compliant_tag});
}

} // namespace

#endif
